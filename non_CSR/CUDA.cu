#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
using namespace std;

#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 


void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

void construct_matrices(int *n_ptr, int *m_ptr, int *l_ptr,
                        int **a_mat_ptr, int **b_mat_ptr) {

    *a_mat_ptr = new int[*n_ptr * *m_ptr];
    *b_mat_ptr = new int[*m_ptr * *l_ptr];

    for (int i = 0; i < *n_ptr; ++i) {
        for (int j = 0; j < *m_ptr; ++j) {
            std::cin >> (*a_mat_ptr)[i * *m_ptr + j];
        }
    }

    for (int i = 0; i < *m_ptr; ++i) {
        for (int j = 0; j < *l_ptr; ++j) {
            std::cin >> (*b_mat_ptr)[i * *l_ptr + j];
        }
    }
}

int main(int argc, char const *argv[])
{
    ios_base::sync_with_stdio(false);
    cin.tie(0);
    int num;
    cin >> num;
    for(int i = 0; i < num; i++){
        int m, n, k;
        srand(3333);
        cin >> m >> n >> k;
        cout << m << n << k;

        float gpu_total_time;
        hipEvent_t start_total, stop_total;
        hipEventCreate(&start_total);
        hipEventCreate(&stop_total);
        hipEventRecord(start_total,0);

        float read_time;
        hipEvent_t start_read, stop_read;
        hipEventCreate(&start_read);
        hipEventCreate(&stop_read);

        

        int *h_a, *h_b, *h_c, *h_cc;
        hipHostMalloc((void **) &h_a, sizeof(int)*m*n);
        hipHostMalloc((void **) &h_b, sizeof(int)*n*k);
        hipHostMalloc((void **) &h_c, sizeof(int)*m*k);
        hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);

        hipEventRecord(start_read,0);
        construct_matrices(&m, &n, &k, &h_a, &h_b);
        hipEventRecord(stop_read, 0);
        hipEventSynchronize(stop_read);


        float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        
        int *d_a, *d_b, *d_c;
        hipMalloc((void **) &d_a, sizeof(int)*m*n);
        hipMalloc((void **) &d_b, sizeof(int)*n*k);
        hipMalloc((void **) &d_c, sizeof(int)*m*k);

        hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

        unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
        // Launch kernel 
        if(m == n && n == k)
        {
            // gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);    
        }
        else
        {
            hipEventRecord(start, 0);
            gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);   
            hipEventRecord(stop, 0);

        }

        hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        hipEventSynchronize(stop);
        hipEventSynchronize(stop_total);
        hipEventRecord(stop_total, 0);


        // compute time elapse on GPU computing
        hipEventElapsedTime(&read_time, start_read, stop_read);
        printf("Read time: %f ms.",  read_time);

        hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
        printf("Mul time: %f ms.",  gpu_elapsed_time_ms);

        hipEventElapsedTime(&gpu_total_time, start_total, stop_total);
        printf("Total time: %f ms.\n",  gpu_total_time);

        hipEventDestroy(start_read);
        hipEventDestroy(stop_read);
        hipEventDestroy(start_total);
        hipEventDestroy(stop_total);
/*
        // start the CPU version
        hipEventRecord(start, 0);

        cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
        printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n", m, n, n, k, cpu_elapsed_time_ms);

        // validate results computed by GPU
        int all_ok = 1;
        for (int i = 0; i < m; ++i)
        {
            for (int j = 0; j < k; ++j)
            {
                //printf("[%d][%d]:%d == [%d][%d]:%d, ", i, j, h_cc[i*k + j], i, j, h_c[i*k + j]);
                if(h_cc[i*k + j] != h_c[i*k + j])
                {
                    all_ok = 0;
                }
            }
            //printf("\n");
        }

        // roughly compute speedup
        if(all_ok)
        {
            printf("all results are correct!!!, speedup = %f\n\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
        }
        else
        {
            printf("incorrect results\n");
        }*/

        // free memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
        hipHostFree(h_cc);
    }
    return 0;
}